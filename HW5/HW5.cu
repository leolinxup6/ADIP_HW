#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#define _USE_MATH_DEFINES

#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <Windows.h>
#include <ctime>
#include <opencv2\opencv.hpp>

#include "hip/hip_runtime.h"
#include ""


using namespace std;
using namespace cv;

void openRaw(const char filename[], unsigned char* tmp, int h, int w);
void storeRaw(const char filename[], unsigned char* tmp, int h, int w, int size);
double MSE(unsigned char* image1, unsigned char* image2, int h, int w);
double PSNR(unsigned char* image1, unsigned char* image2, int h, int w);
int DFT_phase(unsigned char* phase, double* re, double* im, int h, int w);
int DFT_2D(unsigned char* img, unsigned char* dst, unsigned char* phase, int h, int w);
__global__ void dft_2d_compute(unsigned char* src, double* out, double* re, double* im, int h, int w);
int fit_image(unsigned char* dst, double* out, int h, int w);
int opencv_dft(unsigned char* src, Mat& dst, int h, int w);
int opencv_phase(unsigned char* src, Mat& ph, int h, int w);
int IDFT(unsigned char* src, unsigned char* dst, int h, int w);
__global__ void idft_2d_compute(double* src, double* out, double* re, double* im, int h, int w);
int main()
{
	LARGE_INTEGER m_nFreq;
	LARGE_INTEGER m_nBeginTime;
	LARGE_INTEGER nEndTime;
	double time[15];
	char rect_filename[50] = "rect256.raw";
	char circle_filename[50] = "circle256.raw";
	char square_filename[50] = "Square256.raw";
	char square_rota_filename[50] = "square256_rota.raw";
	const int height = 256, width = 256, size = 256 * 256;
	unsigned char* rect = new unsigned char[size]();
	unsigned char* circle = new unsigned char[size]();
	unsigned char* square = new unsigned char[size]();
	unsigned char* square_rota = new unsigned char[size]();
	unsigned char* rect_phase = new unsigned char[size]();
	unsigned char* circle_phase = new unsigned char[size]();
	unsigned char* square_phase = new unsigned char[size]();
	unsigned char* square_rota_phase = new unsigned char[size]();
	unsigned char* rect_out = new unsigned char[size]();
	unsigned char* circle_out = new unsigned char[size]();
	unsigned char* square_out = new unsigned char[size]();
	unsigned char* square_rota_out = new unsigned char[size]();
	unsigned char* rect_IDFT = new unsigned char[size]();
	unsigned char* circle_IDFT = new unsigned char[size]();
	unsigned char* square_IDFT = new unsigned char[size]();
	unsigned char* square_rota_IDFT = new unsigned char[size]();
	Mat rect_opencv_out, circle_opencv_out, square_opencv_out, square_rota_opencv_out;
	openRaw(rect_filename, rect, height, width);
	openRaw(circle_filename, circle, height, width);
	openRaw(square_filename, square, height, width);
	openRaw(square_rota_filename, square_rota, height, width);

	Mat rect_mat = Mat(height, width, CV_8UC1, rect);
	Mat circle_mat = Mat(height, width, CV_8UC1, circle);
	Mat square_mat = Mat(height, width, CV_8UC1, square);
	Mat square_rota_mat = Mat(height, width, CV_8UC1, square_rota);
	imwrite("rect.png", rect_mat);
	imwrite("circle.png", circle_mat);
	imwrite("square.png", square_mat);
	imwrite("square_rota.png", square_rota_mat);

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	DFT_2D(rect, rect_out, rect_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[0] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat rect_DFT_mat = Mat(height, width, CV_8UC1, rect_out);
	Mat rect_phase_mat = Mat(height, width, CV_8UC1, rect_phase);
	imwrite("rect_DFT.png", rect_DFT_mat);
	imwrite("rect_phase.png", rect_phase_mat);
	//cout << time[0] << "  s" << endl;

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	DFT_2D(circle, circle_out, circle_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[1] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat circle_DFT_mat = Mat(height, width, CV_8UC1, circle_out);
	Mat circle_phase_mat = Mat(height, width, CV_8UC1, circle_phase);
	imwrite("circle_DFT.png", circle_DFT_mat);
	imwrite("circle_phase.png", circle_phase_mat);

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	DFT_2D(square, square_out, square_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[2] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat square_DFT_mat = Mat(height, width, CV_8UC1, square_out);
	Mat square_phase_mat = Mat(height, width, CV_8UC1, square_phase);
	imwrite("square_DFT.png", square_DFT_mat);
	imwrite("square_phase.png", square_phase_mat);

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	DFT_2D(square_rota, square_rota_out, square_rota_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[3] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat square_rota_DFT_mat = Mat(height, width, CV_8UC1, square_rota_out);
	Mat square_rota_phase_mat = Mat(height, width, CV_8UC1, square_rota_phase);
	imwrite("square_rota_DFT.png", square_rota_DFT_mat);
	imwrite("square_rota_phase.png", square_rota_phase_mat);

	//opencv

	Mat rect_opencv_phase;
	Mat circle_opencv_phase;
	Mat square_opencv_phase;
	Mat square_rota_opencv_phase;
	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	opencv_dft(rect, rect_opencv_out, height, width);
	opencv_phase(rect, rect_opencv_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[4] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	imwrite("rect_DFT_opencv.png", rect_opencv_out);
	imwrite("rect_phase_opencv.png", rect_opencv_phase);
	//cout << time[4] << "  s" << endl;

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	opencv_dft(circle, circle_opencv_out, height, width);
	opencv_phase(circle, circle_opencv_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[5] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	imwrite("circle_DFT_opencv.png", circle_opencv_out);
	imwrite("circle_phase_opencv.png", circle_opencv_phase);

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	opencv_dft(square, square_opencv_out, height, width);
	opencv_phase(square, square_opencv_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[6] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	imwrite("square_DFT_opencv.png", square_opencv_out);
	imwrite("square_phase_opencv.png", square_opencv_phase);

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	opencv_dft(square_rota, square_rota_opencv_out, height, width);
	opencv_phase(square_rota, square_rota_opencv_phase, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[7] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	imwrite("square_rota_DFT_opencv.png", square_rota_opencv_out);
	imwrite("square_rota_phase_opencv.png", square_rota_opencv_phase);

	// IDFT

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	IDFT(rect, rect_IDFT, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[8] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat rect_IDFT_mat = Mat(height, width, CV_8UC1, rect_IDFT);
	imwrite("rect_IDFT.png", rect_IDFT_mat);
	//cout << time[8] << "  s" << endl;

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	IDFT(circle, circle_IDFT, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[9] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat circle_IDFT_mat = Mat(height, width, CV_8UC1, circle_IDFT);
	imwrite("circle_IDFT.png", circle_IDFT_mat);

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	IDFT(square, square_IDFT, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[10] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat square_IDFT_mat = Mat(height, width, CV_8UC1, square_IDFT);
	imwrite("square_IDFT.png", square_IDFT_mat);

	QueryPerformanceFrequency(&m_nFreq); // �������P��
	QueryPerformanceCounter(&m_nBeginTime); // �p�ɶ}�l
	IDFT(square_rota, square_rota_IDFT, height, width);
	QueryPerformanceCounter(&nEndTime);// �p�ɺI��
	time[11] = (double)(nEndTime.QuadPart - m_nBeginTime.QuadPart) / m_nFreq.QuadPart;
	Mat square_rota_IDFT_mat = Mat(height, width, CV_8UC1, square_rota_IDFT);
	imwrite("square_rota_IDFT.png", square_rota_IDFT_mat);

int a = 0, b = 0;
	while (a == 0) {
		cout << "Enter the question number : " << endl << "1. my own DFT" << endl << "2. openCV DFT" << endl << "3. IDFT" << endl << "4. Exit" << endl;
		cin >> b;
		switch (b) {
		case 1: {
			imshow("rect_DFT", rect_DFT_mat);
			waitKey();
			destroyWindow("rect_DFT");
			imshow("rect_phase", rect_phase_mat);
			waitKey();
			destroyWindow("rect_phase");

			imshow("circle_DFT", circle_DFT_mat);
			waitKey();
			destroyWindow("circle_DFT");
			imshow("circle_phase", circle_phase_mat);
			waitKey();
			destroyWindow("circle_phase");

			imshow("square_DFT", square_DFT_mat);
			waitKey();
			destroyWindow("square_DFT");
			imshow("square_phase", square_phase_mat);
			waitKey();
			destroyWindow("square_phase");

			imshow("square_rota_DFT", square_rota_DFT_mat);
			waitKey();
			destroyWindow("square_rota_DFT");
			imshow("square_rota_phase", square_rota_phase_mat);
			waitKey();
			destroyWindow("square_rota_phase");

			cout << "rectangle My DFT times : " << time[0] << "  s" << endl;
			cout << "circle My DFT times : " << time[1] << "  s" << endl;
			cout << "square My DFT times : " << time[2] << "  s" << endl;
			cout << "square rotation My DFT times : " << time[3] << "  s" << endl;
			double times = time[0] + time[1] + time[2] + time[3];
			cout << "total My DFT times : " << times << "  s" << endl << endl;
		}
			  break;
		case 2: {

			imshow("rect_opencv_out", rect_opencv_out);
			waitKey();
			destroyWindow("rect_opencv_out");

			imshow("circle_opencv_out", circle_opencv_out);
			waitKey();
			destroyWindow("circle_opencv_out");

			imshow("square_opencv_out", square_opencv_out);
			waitKey();
			destroyWindow("square_opencv_out");

			imshow("square_rota_opencv_out", square_rota_opencv_out);
			waitKey();
			destroyWindow("square_rota_opencv_out");


			imshow("rect_opencv_phase", rect_opencv_phase);
			waitKey();
			destroyWindow("rect_opencv_phase");

			imshow("circle_opencv_phase", circle_opencv_phase);
			waitKey();
			destroyWindow("circle_opencv_phase");

			imshow("square_opencv_phase", square_opencv_phase);
			waitKey();
			destroyWindow("square_opencv_phase");

			imshow("square_rota_opencv_phase", square_rota_opencv_phase);
			waitKey();
			destroyWindow("square_rota_opencv_phase");

			cout << "My retangle DFT with openCV PSNR : " << PSNR(rect_out, rect_opencv_out.data, height, width) << "  db  MSE : " << MSE(rect_out, rect_opencv_out.data, height, width) << endl;
			cout << "My circle DFT with openCV PSNR : " << PSNR(circle_out, circle_opencv_out.data, height, width) << "  db  MSE : " << MSE(circle_out, circle_opencv_out.data, height, width) << endl;
			cout << "My square DFT with openCV PSNR : " << PSNR(square_out, square_opencv_out.data, height, width) << "  db  MSE : " << MSE(square_out, square_opencv_out.data, height, width) << endl;
			cout << "My square rotation DFT with openCV PSNR : " << PSNR(square_rota_out, square_rota_opencv_out.data, height, width) << "  db  MSE : " << MSE(square_rota_out, square_rota_opencv_out.data, height, width) << endl;

			cout << "My retangle DFT phase with openCV PSNR : " << PSNR(rect_phase, rect_opencv_phase.data, height, width) << "  db  MSE : " << MSE(rect_phase, rect_opencv_phase.data, height, width) << endl;
			cout << "My circle DFT phase with openCV PSNR : " << PSNR(circle_phase, circle_opencv_phase.data, height, width) << "  db  MSE : " << MSE(circle_phase, circle_opencv_phase.data, height, width) << endl;
			cout << "My square DFT phase with openCV PSNR : " << PSNR(square_phase, square_opencv_phase.data, height, width) << "  db  MSE : " << MSE(square_phase, square_opencv_phase.data, height, width) << endl;
			cout << "My square rotation DFT phase with openCV PSNR : " << PSNR(square_rota_phase, square_rota_opencv_phase.data, height, width) << "  db  MSE : " << MSE(square_rota_phase, square_rota_opencv_phase.data, height, width) << endl;

			cout << "rectangle opencv DFT times : " << time[4] << "  s" << endl;
			cout << "circle opencv DFT times : " << time[5] << "  s" << endl;
			cout << "square opencv DFT times : " << time[6] << "  s" << endl;
			cout << "square rotation opencv DFT times : " << time[7] << "  s" << endl;
			double times = time[4] + time[5] + time[6] + time[7];
			cout << "total opencv DFT times : " << times << "  s" << endl << endl;
			//Mat rect_opencv_phase;
			//Mat circle_opencv_phase;
			//Mat square_opencv_phase;
			//Mat square_rota_opencv_phase;
		}
			  break;
		case 3: {

			imshow("rect_IDFT", rect_IDFT_mat);
			waitKey();
			destroyWindow("rect_IDFT");

			imshow("circle_IDFT", circle_IDFT_mat);
			waitKey();
			destroyWindow("circle_IDFT");

			imshow("square_IDFT", square_IDFT_mat);
			waitKey();
			destroyWindow("square_IDFT");

			imshow("square_rota_IDFT", square_rota_IDFT_mat);
			waitKey();
			destroyWindow("square_rota_IDFT");

			cout << "rectangle IDFT with retangle PSNR : " << PSNR(rect, rect_IDFT, height, width) << "  db  MSE : " << MSE(rect, rect_IDFT, height, width) << endl;
			cout << "circle IDFT with circle PSNR : " << PSNR(rect, rect_IDFT, height, width) << "  db  MSE : " << MSE(rect, rect_IDFT, height, width) << endl;
			cout << "square IDFT with square PSNR : " << PSNR(rect, rect_IDFT, height, width) << "  db  MSE : " << MSE(rect, rect_IDFT, height, width) << endl;
			cout << "square rotation IDFT with square rotation PSNR : " << PSNR(rect, rect_IDFT, height, width) << "  db  MSE : " << MSE(rect, rect_IDFT, height, width) << endl;

			cout << "rectangle IDFT times : " << time[8] << "  s" << endl;
			cout << "circle IDFT times : " << time[9] << "  s" << endl;
			cout << "square IDFT times : " << time[10] << "  s" << endl;
			cout << "square rotation IDFT times : " << time[11] << "  s" << endl;
			double times = time[8] + time[9] + time[10] + time[11];
			cout << "total IDFT times : " << times << "  s" << endl << endl;
		}
			  break;
		case 4: {
			a = 1;
		}
			  break;
		default:
			break;
		}
	}
	return 0;
	
}
int DFT_phase(unsigned char* phase, double* re, double* im, int h, int w) {
	const int size = h * w;
	double* temp = new double[size];
	double a = 0;
	for (int i = 0; i < size; i++) {
		
		a = atan2(im[i], re[i]);
		temp[i] = a;
	}


	fit_image(phase, temp, h, w);
	return 1;
}
int opencv_phase(unsigned char* src, Mat& ph, int h, int w) {
	//test phase
	Mat img, pad12, complexI12;
	img = Mat(h, w, CV_8UC1, src);
	int Rr = getOptimalDFTSize(img.rows);
	int Cc = getOptimalDFTSize(img.cols);
	copyMakeBorder(img, pad12, 0, Rr - img.rows, 0, Cc - img.cols, BORDER_CONSTANT, Scalar::all(0));
	Mat planesph[] = { Mat_<float>(pad12), Mat::zeros(pad12.size(), CV_32F) };
	merge(planesph, 2, complexI12);
	dft(complexI12, complexI12);
	split(complexI12, planesph);
	phase(planesph[0], planesph[1], ph, false);
	int a = ph.cols / 2;
	int b = ph.rows / 2;
	Mat temp;
	Mat c0(ph, Rect(0, 0, a, b));
	Mat c1(ph, Rect(a, 0, a, b));
	Mat c2(ph, Rect(0, b, a, b));
	Mat c3(ph, Rect(a, b, a, b));
	c0.copyTo(temp);
	c3.copyTo(c0);
	temp.copyTo(c3);
	c1.copyTo(temp);
	c2.copyTo(c1);
	temp.copyTo(c2);
	normalize(ph, ph, 0, 1, NORM_MINMAX);
	ph.convertTo(ph, CV_8UC1, 255, 0);
	//end test
	
	return 1;
}

int opencv_dft(unsigned char* src, Mat& dst, int h, int w) {
	Mat imgi, imgo, pad, complexI;
	imgi = Mat(h, w, CV_8UC1, src);
	int R = getOptimalDFTSize(imgi.rows);
	int C = getOptimalDFTSize(imgi.cols);
	copyMakeBorder(imgi, pad, 0, R - imgi.rows, 0, C - imgi.cols, BORDER_CONSTANT, Scalar::all(0));
	Mat planes[] = { Mat_<float>(pad), Mat::zeros(pad.size(), CV_32F) };
	merge(planes, 2, complexI);
	dft(complexI, complexI);
	split(complexI, planes);
	magnitude(planes[0], planes[1], planes[0]);
	imgo = planes[0];
	imgo += Scalar::all(1);
	log(imgo, imgo);
	imgo = imgo(Rect(0, 0, imgo.cols & -2, imgo.rows & -2));

	int x = imgo.cols / 2;
	int y = imgo.rows / 2;
	Mat tmp;
	Mat q0(imgo, Rect(0, 0, x, y));
	Mat q1(imgo, Rect(x, 0, x, y));
	Mat q2(imgo, Rect(0, y, x, y));
	Mat q3(imgo, Rect(x, y, x, y));
	q0.copyTo(tmp);
	q3.copyTo(q0);
	tmp.copyTo(q3);
	q1.copyTo(tmp);
	q2.copyTo(q1);
	tmp.copyTo(q2);
	normalize(imgo, imgo, 0, 1, NORM_MINMAX);
	imgo.convertTo(imgo, CV_8UC1, 255, 0);
	imgo.copyTo(dst);

	return 1;
}
int IDFT(unsigned char* src, unsigned char* dst, int h, int w) {
	const int size = h * w;
	double* re = new double[size];
	double* im = new double[size];
	double* out = new double[size];
	double* idft = new double[size];

	dim3 grid(8, 8);
	dim3 thread(32, 32);
	unsigned char* d_src;
	double* d_re;
	double* d_im;
	double* d_out;
	double* d_idft;
	size_t size1 = size * sizeof(uchar);
	size_t size2 = size * sizeof(double);
	hipMalloc((void**)&d_src, size1);
	hipMemcpy(d_src, src, size1, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_re, size2);
	hipMalloc((void**)&d_im, size2);
	hipMalloc((void**)&d_out, size2);
	hipMalloc((void**)&d_idft, size2);
	dft_2d_compute << <grid, thread >> > (d_src, d_out, d_re, d_im, h, w);
	idft_2d_compute << <grid, thread >> > (d_out, d_idft, d_re, d_im, h, w);
	hipMemcpy(idft, d_idft, size2, hipMemcpyDeviceToHost);
	fit_image(dst, idft, h, w);
	hipFree(d_src);
	hipFree(d_re);
	hipFree(d_im);
	hipFree(d_out);
	hipFree(d_idft);
	return 1;

}
int DFT_2D(unsigned char* src, unsigned char* dst, unsigned char* phase, int h, int w) {
	const int size = h * w;
	double* re = new double[size];
	double* im = new double[size];
	double* out = new double[size];

	dim3 grid(8, 8);
	dim3 thread(32, 32);
	unsigned char* d_src;
	double* d_re;
	double* d_im;
	double* d_out;
	size_t size1 = size * sizeof(uchar);
	size_t size2 = size * sizeof(double);
	hipMalloc((void**)&d_src, size1);
	hipMemcpy(d_src, src, size1, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_re, size2);
	hipMalloc((void**)&d_im, size2);
	hipMalloc((void**)&d_out, size2);
	dft_2d_compute << <grid, thread >> > (d_src, d_out, d_re, d_im, h, w);
	hipMemcpy(re, d_re, size2, hipMemcpyDeviceToHost);
	hipMemcpy(im, d_im, size2, hipMemcpyDeviceToHost);
	hipMemcpy(out, d_out, size2, hipMemcpyDeviceToHost);
	fit_image(dst, out, h, w);
	DFT_phase(phase,re,im,h,w);
	hipFree(d_src);
	hipFree(d_re);
	hipFree(d_im);
	hipFree(d_out);
	return 1;
}

__global__ void idft_2d_compute(double* src, double* out, double* re, double* im, int h, int w) {

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	double real = 0.0, image = 0.0;
	double constant;

	for (int v = 0; v < h; v++) {
		for (int u = 0; u < w; u++) {
			constant = -2 * M_PI * ((u * x * 1.0 / w) + (v * y * 1.0 / h));
			real = real + re[u + v * w] * cos(constant) + im[u + v * w] * sin(constant);
			image = image + im[u + v * w] * cos(constant) - re[u + v * w] * sin(constant);
		}
	}
	double temp = (sqrt(real * real + image * image)) / (sqrt(h * w * 1.0));
	//��J
	out[x + y * w] = (temp);
}
__global__ void dft_2d_compute(unsigned char* src, double* out, double* re, double* im, int h, int w) {
	int u = threadIdx.x + blockIdx.x * blockDim.x;
	int v = threadIdx.y + blockIdx.y * blockDim.y;
	double real = 0.0, image = 0.0;
	double constant;

	for (int y = 0; y < h; y++) {
		for (int x = 0; x < w; x++) {
			constant = 2 * M_PI * ((u * x * 1.0 / w) + (v * y * 1.0 / h));
			real = real + powf(-1, x + y) * src[x + y * w] * cos(constant);
			image = image - powf(-1, x + y) * src[x + y * w] * sin(constant);
		}
	}
	//��J
	out[u + v * w] = log((sqrt(real * real + image * image)) + 1);
	re[u + v * w] = real;
	im[u + v * w] = image;
}
int fit_image(unsigned char* dst, double* out, int h, int w) {
	int size = h * w;
	double max = -100000, min = 10000000;
	for (int i = 0; i < size; i++) {
		if (out[i] > max)
			max = out[i];
		else if (out[i] < min)
			min = out[i];
	}
	for (int i = 0; i < size; i++) {
		out[i] = (out[i] - min) * 255 / (max - min) + 0.5;
	}
	for (int i = 0; i < size; i++) {
		dst[i] = (int)out[i];
	}
	return 1;
}
void openRaw(const char filename[], unsigned char* tmp, int h, int w)
{
	FILE* fp;
	errno_t err;
	int size = h * w;
	err = fopen_s(&fp, filename, "rb");
	if (err == 0)
	{
		fread_s(tmp, size, size, 1, fp);
		fclose(fp);
	}
	else
	{
		cout << "open " << filename << " error!" << endl;
	}
}
void storeRaw(const char filename[], unsigned char* tmp, int h, int w, int size) {
	FILE* resultFile;
	errno_t err;
	err = fopen_s(&resultFile, filename, "wb");

	if (err == 0)
	{
		fwrite(tmp, 1, size, resultFile);
		fclose(resultFile);
		cout << "save " << filename << " success~" << endl;
	}
	else
	{
		cout << "store " << filename << " error!" << endl;
	}
}
double MSE(unsigned char* image1, unsigned char* image2, int h, int w) {
	int sum_sq = 0;
	double mse;

	for (int i = 0; i < h; ++i)
	{
		for (int j = 0; j < w; ++j)
		{
			int p1 = image1[i * w + j];
			int p2 = image2[i * w + j];
			int err = p2 - p1;
			sum_sq += (err * err);
		}
	}
	return mse = (double)sum_sq / (h * w);
}
double PSNR(unsigned char* image1, unsigned char* image2, int h, int w) {
	double mse = 0;
	mse = MSE(image1, image2, h, w);
	return (10 * log10((255 * 255) / mse));
}